#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#include "fft.h"
#include "utils.h"
extern "C"
{
#include "timer.h"
#include "cuda_fft.h"
}

extern "C"
void cuda_fft(complex_t *x, complex_t *x_f, int n, int repetitions, float* time)
{
  
  reset_timer(); 
  double DDD = get_time();
  hipfftHandle plan;
  hipfftResult err = hipfftPlan1d(&plan, n, HIPFFT_Z2Z, 1); 
  
  if (err != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Execution failed, error code is %d\n", err);
    exit(-1); 
  }

  printf("Time to create cuFFT plan: %lf\n", get_time()-DDD);
 
  //hipEvent_t start, stop;
  //hipEventCreate(&start);
  //hipEventCreate(&stop); 

  //hipEventRecord(start);
  
  complex_t *cufft_x_f = (complex_t *)malloc(n*sizeof(complex_t)); 
  complex_t *d_x, *d_x_f;
  
  hipMalloc((void**)&d_x, n*sizeof(complex_t));
  hipMemcpy(d_x, x, n*sizeof(complex_t),hipMemcpyHostToDevice);
  
  hipMalloc((void**)&d_x_f, n*sizeof(complex_t));

  for(int i = 0; i < repetitions; i++){
    err = hipfftExecZ2Z(plan, (hipfftDoubleComplex *)d_x, (hipfftDoubleComplex *)d_x_f, HIPFFT_FORWARD);
    if (err != HIPFFT_SUCCESS){
      fprintf(stderr, "CUFFT error: Execution failed, error code is %d\n", err);
      exit(-1); 
    }
  }
  
  hipMemcpy(cufft_x_f, d_x_f, n*sizeof(complex_t), hipMemcpyDeviceToHost);

  hipFree(d_x);
  hipFree(d_x_f);
  
  //hipEventRecord(stop);
  //hipEventSynchronize(stop);
  //hipEventElapsedTime(time, start, stop);
  //
  //hipEventDestroy(start);
  //hipEventDestroy(stop);
  printf("Time to run cuFFT : %f\n", get_time());
  
  real_t CUFFT_ERROR =0;
  for(int i=0; i< n ; i++){
      CUFFT_ERROR += cabs(cufft_x_f[i]/n- x_f[i]);
  }
  printf("ERROR of CUFFT is %lg\n", CUFFT_ERROR);
  hipfftDestroy(plan);
  free(cufft_x_f);
  
}

extern "C"
void cufft_plan_create(unsigned int* plan, int B, int loops)
{
   //hipFree(0);
   hipfftHandle *plan1 = (hipfftHandle*)plan;
   hipfftResult err;
   err = hipfftPlan1d(plan1, B, HIPFFT_Z2Z, loops); 
   if (err != HIPFFT_SUCCESS){
     fprintf(stderr, "CUFFT error: Plan creation failed");
     exit(-1); 
   }
}
