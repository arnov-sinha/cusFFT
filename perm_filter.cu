#include "hip/hip_runtime.h"
#include "fft.h"
#include "hipfft/hipfft.h"
#include "utils.h"
#include "perm_filter.h"
extern "C"
{
#include "timer.h"
}
#include "hip/hip_complex.h"
#include <hip/hip_runtime.h>

__global__ void PermFilterKernel(hipDoubleComplex* d_origx, hipDoubleComplex* d_filter, int* d_permute, hipDoubleComplex* d_x_sampt, int B, int n, int loops, int round)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i < loops*B)
  {
    int i_mod_B = i & (B-1);
    int off = i_mod_B * round;
    int tmp = i/B;
    int ai = d_permute[tmp];
    hipDoubleComplex tmp_value1, tmp_value2;

    for(int j=0; j<round; j++){
      int index = (i_mod_B + B*j)*ai &(n-1);
      
      tmp_value1 = hipCmul(d_origx[index],d_filter[off+j]);
      tmp_value2 = hipCadd(tmp_value1, tmp_value2);
    }
    d_x_sampt[i] = tmp_value2;
  }
}


extern "C"
void inner_loop_step_a_plus_b(complex_t *origx, Filter *filter, complex_t *x_sampt, int*permute, int n, int B, int loops, 
    double *PF_ALL, double *B_ALL, float *DtoH, float *HtoD, unsigned int plan1)
{
  int filter_size = filter->sizet;
  int round = filter_size/B;
  complex_t *d_origx, *d_filter, *d_x_sampt;
  int *d_permute;
  
  hipfftHandle plan = (hipfftHandle)plan1;
  hipfftResult err;
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //Start of timing on HtoD
  hipEventRecord(start);
  
  //Allocate device memory and copy to device
  hipMalloc((void**)&d_origx, n*sizeof(complex_t));
  hipMemcpy(d_origx, origx, n*sizeof(complex_t), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_filter, filter_size*sizeof(complex_t));
  hipMemcpy(d_filter, filter->time, filter_size*sizeof(complex_t), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_permute, loops*sizeof(int));
  hipMemcpy(d_permute, permute, loops*sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_x_sampt, loops*B*sizeof(complex_t));
 
  //End of timing on HtoD 
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float time;
  hipEventElapsedTime(&time,start, stop);
  *HtoD = (float)time/1e3;

  //Start of the kernels
  //Start of timing on kernels
  hipEventRecord(start);

  dim3 dimBlock(512);
  dim3 dimGrid(loops*B/dimBlock.x);
  PermFilterKernel<<<dimGrid, dimBlock>>>((hipDoubleComplex*)d_origx, (hipDoubleComplex*)d_filter, d_permute, (hipDoubleComplex*)d_x_sampt, B, n,loops,round);
  
  //End of timing on kernerls 
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time,start, stop);
  *PF_ALL = (float)time/1e3;
  
  //Step B -- cuFFT of B-dimensional FFT
  double DDD = get_time();
    
  err = hipfftExecZ2Z(plan, (hipfftDoubleComplex *)d_x_sampt, (hipfftDoubleComplex *)d_x_sampt, HIPFFT_FORWARD);
  if (err != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Execution failed, error code is %d\n", err);
    exit(-1); 
  }
  
  *B_ALL = get_time() - DDD;

  
  //Transfer back the d_x_sampt in freq domain
  //Start of timing on DtoH
  hipEventRecord(start);

  hipMemcpy(x_sampt, d_x_sampt, loops*B*sizeof(complex_t), hipMemcpyDeviceToHost); 
  
  //End of timing on DtoH 
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time,start, stop);
  *DtoH = (float)time/1e3;
  

  //destroy plan and device memory
  hipfftDestroy(plan);
  hipFree(d_origx);
  hipFree(d_filter);
  hipFree(d_x_sampt);
  hipFree(d_permute);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

